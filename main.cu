
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>


typedef unsigned int uint;
typedef unsigned char uchar;

__device__
void cmap(uint value, uchar *pixel){

    pixel[1] = (255-value)*.4;
    pixel[2] = (value%10)*255/10;
    pixel[0] = 255*(1-(255/(255+value)))*.65;

}

__global__
void julia(uchar *img, uint width, double r_from, double i_from, double step, hipDoubleComplex c, uint n_iter, uint MAX){

    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < MAX) {
        uint x = index % width;
        uint y = index / width;
        hipDoubleComplex z = make_hipDoubleComplex(r_from + x * step, i_from + y * step);
        uint iter = 0;
        for (iter = 0; iter < n_iter && hipCabs(hipCmul(z, z)) < 4; iter++) {
            z = hipCadd(hipCmul(z, z), c);
        }
        cmap(iter, &img[index*3]);
    }
}

void savebmp(char *name,uchar *buffer,int x,int y) {
    FILE *f=fopen(name,"wb");
    if(!f) {
        printf("Error writing image to disk.\n");
        return;
    }
    unsigned int size=x*y*3+54;
    uchar header[54]={'B','M',size&255,(size>>8)&255,(size>>16)&255,size>>24,0,
                      0,0,0,54,0,0,0,40,0,0,0,x&255,x>>8,0,0,y&255,y>>8,0,0,1,0,24,0,0,0,0,0,0,
                      0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    fwrite(header,1,54,f);
    fwrite(buffer,1,x*y*3,f);
    fclose(f);
}


int main() {

    // image params
    uint X = 1920;  // resulting image width
    uint Y = 1080;  // resulting image height
    uint N = X*Y;   // total pixels

    // julia params
    hipDoubleComplex c = make_hipDoubleComplex(-.788, .145);
    double r_min = -.18; // real lower bound
    double r_max = -.08; // real upper bound
    double step = (r_max-r_min)/X; // step length
    double i_min = 0.02;  // imaginary lower bound
    uint max_iter = 255;  // max number of iterations per pixel

    // memalloc
    uchar* img = (uchar*)calloc(N*3, sizeof(uchar));
    uchar* d_img;
    hipMalloc(&d_img, N*3*sizeof(uchar));


    julia <<< (N / 1024)+(N%1024==0?0:1), 1024 >>> (d_img, X, r_min, i_min, step, c, max_iter, N);
    hipMemcpy(img, d_img, N * 3 * sizeof(uchar), hipMemcpyDeviceToHost);
    savebmp("img.bmp", img, X, Y);


    return 0;
}

